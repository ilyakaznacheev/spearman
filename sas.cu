
#include <hip/hip_runtime.h>
__global__ void subtract_and_square(float *dest, float *a, float *b, int n)
{
    // const int index = threadIdx.x * (threadIdx.y + 1);
    // dest[index] = ( a[index] - b[index] ) * ( a[index] - b[index] );
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n)
        dest[index] = ( a[index] - b[index] ) * ( a[index] - b[index] );
}