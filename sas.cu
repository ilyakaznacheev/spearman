
#include <hip/hip_runtime.h>
__global__ void subtract_and_square(float *dest, float *a, float *b)
{
    const int index = threadIdx.x * (threadIdx.y + 1);
    dest[index] = ( a[index] - b[index] ) * ( a[index] - b[index] );
}